#include "hip/hip_runtime.h"
#include "kernel.h"
#include "main.h"
#include <cmath>
#include <time.h> 
#include <thrust/copy.h>

__global__ void copy(const int *a, int *b, int length)
{
	int i = threadIdx.x+ (blockIdx.x * blockDim.x);
	if(i<length){
		b[i] = a[i];
	}
}

__global__ void scan(int *b, int d)
{
    int k = threadIdx.x+ (blockIdx.x * blockDim.x);
	int d2 = pow(2.0f,d-1);

    if(k>=d2)
		b[k] = b[k-d2] + b[k];
}



void prefix_sum(const int *a, int *b, int length)
{
	if(length<1)
		return;

	copy<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(a,b,length);
	hipDeviceSynchronize();
	int t= log((float)length)/log(2.0f)+1;
	for (int d =1 ;d<=t;d++){
		scan<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(b,d);
		hipDeviceSynchronize();
	}
}

__global__ void shared_scan_1block(int *b,int t)
{
    int k = threadIdx.x;
	int d2;
	__shared__ int bs[BLOCK_SIZE];

	if(k<BLOCK_SIZE)
		bs[k] = b[k];

	__syncthreads();
	for (int d =1 ;d<=t;d++){
		d2 = pow(2.0f,d-1);
		if(k>=d2)
			bs[k] = bs[k-d2] + bs[k];
		__syncthreads();
	}

	if(k<BLOCK_SIZE)
		b[k] = bs[k];
}

__global__ void shared_scan(int *b,int t)
{
    int k = threadIdx.x;
	int d2;
	__shared__ int bs[BLOCK_SIZE];
	__shared__ int tempSum;
	tempSum = 0;

	for(int i=0;i<=blockIdx.x;i++){
		int index = threadIdx.x+ (blockIdx.x * blockDim.x);
		if(k==0)
			bs[k] = b[index]+tempSum;
		else
			bs[k] = b[index];

		__syncthreads();
		for (int d =1 ;d<=t;d++){
			d2 = pow(2.0f,d-1);
			if(k>=d2)
				bs[k] = bs[k-d2] + bs[k];
			__syncthreads();
		}

		if(i==blockIdx.x)
			b[index] = bs[k];

		if(k == BLOCK_SIZE-1)
			tempSum = bs[k];

	}
}

void prefix_sum_shared(const int *a, int *b, int length)
{
	if(length<1)
		return;

	copy<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(a,b,length);
	int t= log((float)length)/log(2.0f)+1;
	if(length<=BLOCK_SIZE)
		shared_scan_1block<<<1, length>>>(b,t);
	else
		shared_scan<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(b,t);
}

__global__ void transform_to_boolean(const int *a, int *b, int length)
{
	int i = threadIdx.x+ (blockIdx.x * blockDim.x);
	if(i<length){
		b[i] = a[i]>0? 1:0;
	}
}

void scatter(const int *a, int *b, int length)
{
	if(length<1)
		return;

	transform_to_boolean<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(a,b,length);
	int t= log((float)length)/log(2.0f)+1;
	if(length<=BLOCK_SIZE)
		shared_scan_1block<<<1, length>>>(b,t);
	else
		shared_scan<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(b,t);
}

__global__ void compact(const int *a,int *temp, int *b, int length)
{
	int i = threadIdx.x+ (blockIdx.x * blockDim.x);
	if(i<length){
		if(a[i]!=0)
			b[temp[i]-1]=a[i];
	}
}

void stream_compact(const int *a, int *b, int length)
{
	int *temp = 0;
	hipMalloc((void**)&temp, length * sizeof(int));
	scatter(a, temp,length);
	compact<<<length/BLOCK_SIZE, BLOCK_SIZE>>>(a,temp,b,length);
}

struct is_zero
{
   __host__ __device__
   bool operator()(const int x)
   {
     return x != 0;
   }
};


int main(int argc, char** argv)
{
	int size = 0;
	size = atoi(argv[1]); 
	int *a = new int[size];
	//size = 10;
	//int a[] = {0,0,3,4,0,6,6,7,0,1};
	int *b = new int[size];
	for(int i=0;i<size;i++)
		a[i] = 1;
	
	//clock_t time; 
	//time=clock(); 

	int *dev_a = 0;
    int *dev_b = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );	
	
	prefix_sum(dev_a, dev_b, size);
	//serial_prefix_sum(a,b,size);
	//prefix_sum_shared(dev_a,dev_b,size);
	//serial_scatter(a,b,size);
	//stream_compact(dev_a,dev_b,size);
	//thrust::copy_if(a, a+ size, b, is_zero());

    
	//time=clock()-time; 
	hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop ); 
	hipEventElapsedTime( &time, start, stop );
	cout<<time<<endl;
	hipEventDestroy( start ); 
	hipEventDestroy( stop );
    cudaStatus = hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
	//for(int i=0;i<100;i++)
	//	cout<<b[i]<<endl;
	//cout<<double(time)<<endl; 

    return 0;
}
